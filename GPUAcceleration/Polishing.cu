#include "hip/hip_runtime.h"
#include "Functions.h"
#include <hip/device_functions.h>
using namespace gtom;

#define SHIFT_THREADS 128

__global__ void PolishingGetDiffKernel(float2* d_phase, float2* d_average, float2* d_shiftfactors, float2* d_ctfcoords, CTFParamsLean* d_ctfparams, float* d_invsigma, uint length, float2* d_shifts, float* d_diff, float* d_debugdiff);


__declspec(dllexport) void CreatePolishing(float* d_particles, float2* d_particlesft, float* d_masks, int2 dims, int2 dimscropped, int nparticles, int nframes)
{
	float* d_temp;
	hipMalloc((void**)&d_temp, ElementsFFT2(dims) * nparticles * sizeof(float2));

	for(int z = 0; z < nframes / 3; z++)
	{
		hipMemcpy(d_temp, d_particles + Elements2(dims) * nparticles * (z * 3 + 0), Elements2(dims) * nparticles * sizeof(float), hipMemcpyDeviceToDevice);
		d_AddVector(d_temp, d_particles + Elements2(dims) * nparticles * (z * 3 + 1), d_temp, Elements2(dims) * nparticles);
		d_AddVector(d_temp, d_particles + Elements2(dims) * nparticles * (z * 3 + 2), d_temp, Elements2(dims) * nparticles);

		//d_NormMonolithic(d_temp, d_temp, Elements2(dims), d_masks, T_NORM_MEAN01STD, nparticles);
	    //d_MultiplyByVector(d_temp, d_masks, d_temp, Elements2(dims) * nparticles);
		float radius = 90.0f / (1.0605f / 1.25f);
		d_SphereMask(d_temp, d_temp, toInt3(dims), &radius, 24, NULL, nparticles);
		d_RemapFull2FullFFT(d_temp, d_temp, toInt3(dims), nparticles);
		d_FFTR2C(d_temp, (float2*)d_temp, 2, toInt3(dims), nparticles);
		d_FFTCrop((float2*)d_temp, d_particlesft + ElementsFFT2(dimscropped) * nparticles * z, toInt3(dims), toInt3(dimscropped), nparticles);
	}

	hipFree(d_temp);
}

__declspec(dllexport) void PolishingGetDiff(float2* d_phase, 
												float2* d_average, 
												float2* d_shiftfactors, 
												float2* d_ctfcoords,
												CTFParams* h_ctfparams,
												float* d_invsigma,
												int2 dims, 
												float2* d_shifts,
												float* h_diff, 
												float* h_diffall,
												uint npositions, 
												uint nframes)
{
	int TpB = SHIFT_THREADS;
	dim3 grid = dim3(1, npositions, nframes);

	float* d_diff;
	hipMalloc((void**)&d_diff, npositions * nframes * grid.x * sizeof(float));
	float* d_diffreduced;
	hipMalloc((void**)&d_diffreduced, npositions * nframes * sizeof(float));

	CTFParamsLean* h_lean = (CTFParamsLean*)malloc(npositions * nframes * sizeof(CTFParamsLean));
	for (int i = 0; i < npositions * nframes; i++)
		h_lean[i] = CTFParamsLean(h_ctfparams[i], toInt3(dims));
	CTFParamsLean* d_lean = (CTFParamsLean*)CudaMallocFromHostArray(h_lean, npositions * nframes * sizeof(CTFParamsLean));
	free(h_lean);

	float* d_debugdiff = NULL;
	//hipMalloc((void**)&d_debugdiff, npositions * nframes * ElementsFFT2(dims) * sizeof(float));

	PolishingGetDiffKernel <<<grid, TpB>>> (d_phase, d_average, d_shiftfactors, d_ctfcoords, d_lean, d_invsigma, ElementsFFT2(dims), d_shifts, d_diff, d_debugdiff);

	//d_WriteMRC(d_debugdiff, toInt3(dims.x / 2 + 1, dims.y, npositions * nframes), "d_debugdiff.mrc");

	//d_SumMonolithic(d_diff, d_diffreduced, grid.x, npositions * nframes);
	d_ReduceMean(d_diff, d_diffreduced, npositions, nframes);
	hipMemcpy(h_diff, d_diffreduced, npositions * sizeof(float), hipMemcpyDeviceToHost);

	hipMemcpy(h_diffall, d_diff, npositions * nframes * sizeof(float), hipMemcpyDeviceToHost);
	
	hipFree(d_lean);
	hipFree(d_diffreduced);
	hipFree(d_diff);
}

/*__global__ void PolishingGetDiffKernel(float2* d_phase, float2* d_average, float2* d_shiftfactors, float2* d_ctfcoords, CTFParamsLean* d_ctfparams, float* d_invsigma, uint length, float2* d_shifts, float* d_diff, float* d_debugdiff)
{
	__shared__ float s_diff[SHIFT_THREADS];
	s_diff[threadIdx.x] = 0.0f;

	uint specid = blockIdx.z * gridDim.y + blockIdx.y;
	d_phase += specid * length;
	d_average += specid * length;
	d_debugdiff += specid * length;

	float2 shift = d_shifts[specid];
	float diffsum = 0.0f;

	CTFParamsLean ctfparams = d_ctfparams[specid];

	for (uint id = threadIdx.x; 
		 id < length; 
		 id += SHIFT_THREADS)
	{
		float2 value = d_phase[id];
		float2 average = d_average[id];
		float ctf = d_GetCTF<false>(d_ctfcoords[id].x, d_ctfcoords[id].y, ctfparams);	// Already corrected for mag anisotropy.
		average *= ctf;

		float2 shiftfactors = d_shiftfactors[id];

		float phase = shiftfactors.x * shift.x + shiftfactors.y * shift.y;
		float2 change = make_float2(__cosf(phase), __sinf(phase));
		value = hipCmulf(value, change);

		float2 diff = value - average;

		diffsum += (diff.x * diff.x + diff.y * diff.y) * d_invsigma[id];
		//d_debugdiff[id] = shift.x;
	}

	s_diff[threadIdx.x] = diffsum;
	__syncthreads();

	for (uint lim = 64; lim > 1; lim >>= 1)
	{
		if (threadIdx.x < lim)
		{
			diffsum += s_diff[threadIdx.x + lim];
			s_diff[threadIdx.x] = diffsum;
		}
		__syncthreads();
	}

	if (threadIdx.x == 0)
	{
		diffsum += s_diff[1];

		d_diff[specid] = diffsum / (float)length;
	}
}*/

__global__ void PolishingGetDiffKernel(float2* d_phase, float2* d_average, float2* d_shiftfactors, float2* d_ctfcoords, CTFParamsLean* d_ctfparams, float* d_invsigma, uint length, float2* d_shifts, float* d_diff, float* d_debugdiff)
{
	__shared__ float s_num[SHIFT_THREADS];
	s_num[threadIdx.x] = 0.0f;
	__shared__ float s_denom1[SHIFT_THREADS];
	s_denom1[threadIdx.x] = 0.0f;
	__shared__ float s_denom2[SHIFT_THREADS];
	s_denom2[threadIdx.x] = 0.0f;

	uint specid = blockIdx.z * gridDim.y + blockIdx.y;
	d_phase += specid * length;
	d_average += specid * length;
	d_debugdiff += specid * length;

	float2 shift = d_shifts[specid];
	float numsum = 0.0f, denomsum1 = 0.0f, denomsum2 = 0.0f;

	CTFParamsLean ctfparams = d_ctfparams[specid];

	for (uint id = threadIdx.x; 
		 id < length; 
		 id += SHIFT_THREADS)
	{
		float2 value = d_phase[id];
		float2 average = d_average[id];
		float ctf = d_GetCTF<false>(d_ctfcoords[id].x, d_ctfcoords[id].y, ctfparams);	// Already corrected for mag anisotropy.
		average *= ctf;

		float2 shiftfactors = d_shiftfactors[id];

		float phase = shiftfactors.x * shift.x + shiftfactors.y * shift.y;
		float2 change = make_float2(__cosf(phase), __sinf(phase));
		value = hipCmulf(value, change);

		float invsigma = d_invsigma[id];
		value *= invsigma;
		average *= invsigma;

		numsum += value.x * average.x + value.y * average.y;
		denomsum1 += dotp2(value, value);
		denomsum2 += dotp2(average, average);
	}
	
	s_num[threadIdx.x] = numsum;
	s_denom1[threadIdx.x] = denomsum1;
	s_denom2[threadIdx.x] = denomsum2;
	__syncthreads();

	for (uint lim = 64; lim > 1; lim >>= 1)
	{
		if (threadIdx.x < lim)
		{
			numsum += s_num[threadIdx.x + lim];
			s_num[threadIdx.x] = numsum;
			
			denomsum1 += s_denom1[threadIdx.x + lim];
			s_denom1[threadIdx.x] = denomsum1;
			
			denomsum2 += s_denom2[threadIdx.x + lim];
			s_denom2[threadIdx.x] = denomsum2;
		}
		__syncthreads();
	}

	if (threadIdx.x == 0)
	{
		numsum += s_num[1];
		denomsum1 += s_denom1[1];
		denomsum2 += s_denom2[1];

		d_diff[specid * gridDim.x] = numsum / tmax(1e-6f, sqrt(denomsum1 * denomsum2));
	}
}