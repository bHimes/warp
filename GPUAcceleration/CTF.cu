#include "hip/hip_runtime.h"
#include "Functions.h"
using namespace gtom;

__global__ void ScaleNormCorrSumKernel(half2* d_simcoords, half* d_sim, half* d_scale, half* d_target, CTFParamsLean* d_params, float* d_scores, uint length);

/*

Supplied with a stack of frames, and extraction positions for sub-regions, this method 
extracts portions of each frame, computes the FT, and averages the results as follows:

-3D full fitting: d_output contains all individual spectra from each frame
-2D spatial fitting: d_output contains averages for all positions over all frames
-1D temporal fitting: d_output contains averages for all frames over all positions
-0D no fitting: d_output is NULL

*/

__declspec(dllexport) void CreateSpectra(float* d_frame, 
										int2 dimsframe, 
										int nframes, 
										int3* h_origins, 
										int norigins, 
										int2 dimsregion, 
										int3 ctfgrid, 
										float* d_outputall,
										float* d_outputmean)
{

	int3* d_origins = (int3*)CudaMallocFromHostArray(h_origins, norigins * sizeof(int3));
	tfloat* d_tempspectra;
	hipMalloc((void**)&d_tempspectra, tmax(norigins, nframes) * ElementsFFT2(dimsregion) * sizeof(tfloat));
	tfloat* d_tempaverages;
	hipMalloc((void**)&d_tempaverages, nframes * ElementsFFT2(dimsregion) * sizeof(tfloat));

	bool ctfspace = ctfgrid.x * ctfgrid.y > 1;
	bool ctftime = ctfgrid.z > 1;
	int nspectra = (ctfspace || ctftime) ? (ctfspace ? norigins : 1) * (ctftime ? ctfgrid.z : 1) : 1;

	int pertimegroup = nframes / ctfgrid.z;

	// Temp spectra will be summed up to be averaged later
	d_ValueFill(d_outputall, ElementsFFT2(dimsregion) * nspectra, 0.0f);

	for (int z = 0; z < nframes; z++)
	{
		// Full precision, just write everything to output which is big enough
		//if (ctfspace && ctftime)
		//{
		//	d_CTFPeriodogram(d_frame + Elements2(dimsframe) * z, dimsframe, d_origins, norigins, dimsregion, dimsregion, d_outputall + ElementsFFT2(dimsregion) * norigins * z, false);			
		//	d_AddScalar(d_outputall + ElementsFFT2(dimsregion) * norigins * z, d_outputall + ElementsFFT2(dimsregion) * norigins * z, ElementsFFT2(dimsregion) * norigins, 1e2f);
		//	d_Log(d_outputall + ElementsFFT2(dimsregion) * norigins * z, d_outputall + ElementsFFT2(dimsregion) * norigins * z, ElementsFFT2(dimsregion) * norigins);
		//}
		//else // Partial or no precision
		{
			int framegroup = z / pertimegroup;
			if (framegroup >= ctfgrid.z)
				break;

			// Write spectra to temp and reduce them to a temporary average spectrum
			d_CTFPeriodogram(d_frame + Elements2(dimsframe) * z, dimsframe, d_origins, norigins, dimsregion, dimsregion, d_tempspectra, false);
			d_AddScalar(d_tempspectra, d_tempspectra, ElementsFFT2(dimsregion) * norigins, 1e2f);
			d_Log(d_tempspectra, d_tempspectra, ElementsFFT2(dimsregion) * norigins);

			//d_WriteMRC(d_tempspectra, toInt3(dimsregion.x / 2 + 1, dimsregion.y, norigins), "d_tempspectra.mrc");

			d_ReduceMean(d_tempspectra, d_tempaverages + ElementsFFT2(dimsregion) * z, ElementsFFT2(dimsregion), norigins);

			// Spatially resolved, add to output which has norigins spectra
			//if (ctfspace)
			{
				d_AddVector(d_outputall + ElementsFFT2(dimsregion) * norigins * framegroup, d_tempspectra, d_outputall + ElementsFFT2(dimsregion) * norigins * framegroup, ElementsFFT2(dimsregion) * norigins);
			}
			// Temporally resolved, each spectrum will be the average of the entire frame's spectra (= temporary average, so just copy)
			//else if (ctftime)
			//{
			//	hipMemcpy(d_outputall + ElementsFFT2(dimsregion) * z, d_tempaverages + ElementsFFT2(dimsregion) * z, ElementsFFT2(dimsregion) * sizeof(float), hipMemcpyDeviceToDevice);
			//}
		}
	}

	// Just average over all individual spectra in d_outputall
	//if (ctfspace && ctftime)
		d_DivideByScalar(d_outputall, d_outputall, ElementsFFT2(dimsregion) * Elements(ctfgrid), (tfloat)pertimegroup);
		//d_ReduceMean(d_outputall, d_outputmean, ElementsFFT2(dimsregion), Elements(ctfgrid));
	//else
	//{
	//	// Average output is average of temporary averages
		d_ReduceMean(d_tempaverages, d_outputmean, ElementsFFT2(dimsregion), nframes);
	//
	//	// Those were summed up, so divide by number of summands
	//	if (ctfspace)

	//}
	//d_WriteMRC(d_outputmean, toInt3FFT(dimsregion), "d_outputmean.mrc");
	
	// 0D case, only one average spectrum in outputall
	//if (nspectra == 1)
	//	hipMemcpy(d_outputall, d_outputmean, ElementsFFT2(dimsregion) * sizeof(float), hipMemcpyDeviceToDevice);

	hipFree(d_origins);
	hipFree(d_tempspectra);
	hipFree(d_tempaverages);
}

__declspec(dllexport) CTFParams CTFFitMean(float* d_ps, float2* d_pscoords, int2 dims, CTFParams startparams, CTFFitParams fp, bool doastigmatism)
{
	std::vector<std::pair<tfloat, CTFParams>> fits;
	tfloat score;
	tfloat scoremean;
	tfloat scorestd;

	d_CTFFit(d_ps, d_pscoords, dims, &startparams, 1, fp, 2, fits, score, scoremean, scorestd);

	CTFParams result;
	for (int i = 0; i < 12; i++)
		((tfloat*)&result)[i] = ((tfloat*)&startparams)[i] + ((tfloat*)&(fits[0].second))[i];

	result.Bfactor = score;

	return result;
}

__declspec(dllexport) void CTFMakeAverage(float* d_ps, float2* d_pscoords, uint length, uint sidelength, CTFParams* h_sourceparams, CTFParams targetparams, uint minbin, uint maxbin, int* h_consider, uint batch, float* d_output)
{
	uint nbins = maxbin - minbin;
	if (batch > 1)
		d_CTFRotationalAverageToTarget((tfloat*)d_ps, d_pscoords, length, sidelength, h_sourceparams, targetparams, d_output, minbin, maxbin, h_consider, batch);
	else
		d_CTFRotationalAverageToTarget((tfloat*)d_ps, d_pscoords, length, sidelength, h_sourceparams, targetparams, d_output, minbin, maxbin, NULL, 1);
}

__declspec(dllexport) void CTFCompareToSim(half* d_ps, half2* d_pscoords, half* d_scale, uint length, CTFParams* h_sourceparams, float* h_scores, uint batch)
{
	half* d_sim;
	hipMalloc((void**)&d_sim, length * batch * sizeof(float));
	float* d_scores;
	hipMalloc((void**)&d_scores, batch * sizeof(float));

	CTFParamsLean* h_lean;
	hipHostMalloc((void**)&h_lean, batch * sizeof(CTFParamsLean));
	#pragma omp parallel for
	for (int i = 0; i < batch; i++)
		h_lean[i] = CTFParamsLean(h_sourceparams[i], toInt3(1, 1, 1));	// Sidelength and pixelsize are already included in d_addresses
	CTFParamsLean* d_lean = (CTFParamsLean*)CudaMallocFromHostArray(h_lean, batch * sizeof(CTFParamsLean));
	hipHostFree(h_lean);

	//d_CTFSimulate(h_sourceparams, d_pscoords, d_sim, length, true, batch);

	int TpB = 128;
	dim3 grid = dim3(batch, 1, 1);
	ScaleNormCorrSumKernel <<<grid, TpB>>> (d_pscoords, d_sim, d_scale, d_ps, d_lean, d_scores, length);

	//d_MultiplyByVector(d_sim, d_scale, d_sim, length, batch);
	//d_NormMonolithic(d_sim, d_sim, length, T_NORM_MEAN01STD, batch);
	//d_WriteMRC(d_sim, toInt3(207, 512, 1), "d_sim.mrc");
	//d_WriteMRC(d_ps, toInt3(207, 512, 1), "d_ps.mrc");

	//d_MultiplyByVector(d_ps, d_sim, d_sim, length * batch);

	//d_SumMonolithic(d_sim, d_scores, length, batch);

	hipMemcpy(h_scores, d_scores, batch * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(d_lean);
	hipFree(d_sim);
	hipFree(d_scores);

	//for (uint i = 0; i < batch; i++)
		//h_scores[i] /= (float)length;
}

__global__ void ScaleNormCorrSumKernel(half2* d_simcoords, half* d_sim, half* d_scale, half* d_target, CTFParamsLean* d_params, float* d_scores, uint length)
{
	__shared__ float s_sums1[128];
	__shared__ float s_sums2[128];
	__shared__ float s_mean, s_stddev;

	d_sim += blockIdx.x * length;
	d_target += blockIdx.x * length;

	CTFParamsLean params = d_params[blockIdx.x];

	float sum1 = 0.0, sum2 = 0.0;
	for (uint i = threadIdx.x; i < length; i += blockDim.x)
	{
		float2 simcoords = __half22float2(d_simcoords[i]);
		float pixelsize = params.pixelsize + params.pixeldelta * __cosf(2.0f * (simcoords.y - params.pixelangle));
		simcoords.x /= pixelsize;

		float val = d_GetCTF<true, false>(simcoords.x, simcoords.y, params) * __half2float(d_scale[i]);
		d_sim[i] = __float2half(val);
		sum1 += val;
		sum2 += val * val;
	}
	s_sums1[threadIdx.x] = sum1;
	s_sums2[threadIdx.x] = sum2;
	__syncthreads();

	if (threadIdx.x == 0)
	{
		for (int i = 1; i < 128; i++)
		{
			sum1 += s_sums1[i];
			sum2 += s_sums2[i];
		}

		s_mean = sum1 / (float)length;
		s_stddev = sqrt(((float)length * sum2 - (sum1 * sum1))) / (float)length;
	}
	__syncthreads();

	float mean = s_mean;
	float stddev = s_stddev > 0.0f ? 1.0f / s_stddev : 0.0f;

	sum1 = 0.0f;
	for (uint i = threadIdx.x; i < length; i += blockDim.x)
		sum1 += (__half2float(d_sim[i]) - mean) * stddev * __half2float(d_target[i]);
	s_sums1[threadIdx.x] = sum1;
	__syncthreads();

	if (threadIdx.x == 0)
	{
		for (int i = 1; i < 128; i++)
			sum1 += s_sums1[i];

		d_scores[blockIdx.x] = sum1 / (float)length;
	}
}