#include "hip/hip_runtime.h"
#include "Functions.h"
using namespace gtom;

__declspec(dllexport) void FFT(float* d_input, float2* d_output, int3 dims, uint batch)
{
    d_FFTR2C(d_input, d_output, DimensionCount(dims), dims, batch);
}

__declspec(dllexport) void IFFT(float2* d_input, float* d_output, int3 dims, uint batch)
{
    d_IFFTC2R(d_input, d_output, DimensionCount(dims), dims, batch);
}

__declspec(dllexport) void Pad(float* d_input, float* d_output, int3 olddims, int3 newdims, uint batch)
{
    d_Pad(d_input, d_output, olddims, newdims, T_PAD_VALUE, 0.0f, batch);
}

__declspec(dllexport) void PadFT(float2* d_input, float2* d_output, int3 olddims, int3 newdims, uint batch)
{
    d_FFTPad(d_input, d_output, olddims, newdims, batch);
}

__declspec(dllexport) void CropFT(float2* d_input, float2* d_output, int3 olddims, int3 newdims, uint batch)
{
    d_FFTCrop(d_input, d_output, olddims, newdims, batch);
}

__declspec(dllexport) void RemapToFTComplex(float2* d_input, float2* d_output, int3 dims, uint batch)
{
    d_RemapHalfFFT2Half(d_input, d_output, dims, batch);
}

__declspec(dllexport) void RemapToFTFloat(float* d_input, float* d_output, int3 dims, uint batch)
{
    d_RemapHalfFFT2Half(d_input, d_output, dims, batch);
}

__declspec(dllexport) void RemapFromFTComplex(float2* d_input, float2* d_output, int3 dims, uint batch)
{
    d_RemapHalf2HalfFFT(d_input, d_output, dims, batch);
}

__declspec(dllexport) void RemapFromFTFloat(float* d_input, float* d_output, int3 dims, uint batch)
{
    d_RemapHalf2HalfFFT(d_input, d_output, dims, batch);
}

__declspec(dllexport) void Extract(float* d_input, float* d_output, int3 dims, int3 dimsregion, int3* h_origins, uint batch)
{
	int3* d_origins = (int3*)CudaMallocFromHostArray(h_origins, batch * sizeof(int3));

	d_ExtractMany(d_input, d_output, dims, dimsregion, d_origins, batch);

	hipFree(d_origins);
}

__declspec(dllexport) void ExtractHalf(half* d_input, half* d_output, int3 dims, int3 dimsregion, int3* h_origins, uint batch)
{
	int3* d_origins = (int3*)CudaMallocFromHostArray(h_origins, batch * sizeof(int3));

	d_ExtractMany(d_input, d_output, dims, dimsregion, d_origins, batch);

	hipFree(d_origins);
}

__declspec(dllexport) void ReduceMean(float* d_input, float* d_output, uint vectorlength, uint nvectors, uint batch)
{
	d_ReduceMean(d_input, d_output, vectorlength, nvectors, batch);
}

__declspec(dllexport) void ReduceMeanHalf(half* d_input, half* d_output, uint vectorlength, uint nvectors, uint batch)
{
	d_ReduceMean(d_input, d_output, vectorlength, nvectors, batch);
}

__declspec(dllexport) void Normalize(float* d_ps, float* d_output, uint length, uint batch)
{
	d_NormMonolithic(d_ps, d_output, length, T_NORM_MEAN01STD, batch);
}

__declspec(dllexport) void CreateCTF(float* d_output, float2* d_coords, uint length, CTFParams* h_params, bool amplitudesquared, uint batch)
{
	d_CTFSimulate(h_params, d_coords, d_output, length, amplitudesquared, batch);
}

__declspec(dllexport) void Resize(float* d_input, int3 dimsinput, float* d_output, int3 dimsoutput, uint batch)
{
	d_Scale(d_input, d_output, dimsinput, dimsoutput, T_INTERP_FOURIER, NULL, NULL, batch);
}

__declspec(dllexport) void ShiftStack(float* d_input, float* d_output, int3 dims, float3* h_shifts, uint batch)
{
	d_Shift(d_input, d_output, dims, (tfloat3*)h_shifts, NULL, NULL, NULL, batch);
}

__declspec(dllexport) void Cart2Polar(float* d_input, float* d_output, int2 dims, uint innerradius, uint exclusiveouterradius, uint batch)
{
	d_Cart2Polar(d_input, d_output, dims, T_INTERP_LINEAR, innerradius, exclusiveouterradius, batch);
}

__declspec(dllexport) void Cart2PolarFFT(float* d_input, float* d_output, int2 dims, uint innerradius, uint exclusiveouterradius, uint batch)
{
	d_Cart2PolarFFT(d_input, d_output, dims, T_INTERP_LINEAR, innerradius, exclusiveouterradius, batch);
}

__declspec(dllexport) void Xray(float* d_input, float* d_output, float ndevs, int2 dims, uint batch)
{
    d_Xray(d_input, d_output, toInt3(dims), ndevs, 5, batch);
}

// Arithmetics:

__declspec(dllexport) void Amplitudes(float2* d_input, float* d_output, size_t length)
{
    d_Abs(d_input, d_output, length);
}

__declspec(dllexport) void Sign(float* d_input, float* d_output, size_t length)
{
    d_Sign(d_input, d_output, length);
}

__declspec(dllexport) void AddToSlices(float* d_input, float* d_summands, float* d_output, size_t sliceelements, uint slices)
{
	d_AddVector(d_input, d_summands, d_output, sliceelements, slices);
}

__declspec(dllexport) void SubtractFromSlices(float* d_input, float* d_subtrahends, float* d_output, size_t sliceelements, uint slices)
{
	d_SubtractVector(d_input, d_subtrahends, d_output, sliceelements, slices);
}

__declspec(dllexport) void MultiplySlices(float* d_input, float* d_multiplicators, float* d_output, size_t sliceelements, uint slices)
{
	d_MultiplyByVector(d_input, d_multiplicators, d_output, sliceelements, slices);
}

__declspec(dllexport) void DivideSlices(float* d_input, float* d_divisors, float* d_output, size_t sliceelements, uint slices)
{
	d_DivideByVector(d_input, d_divisors, d_output, sliceelements, slices);
}

__declspec(dllexport) void AddToSlicesHalf(half* d_input, half* d_summands, half* d_output, size_t sliceelements, uint slices)
{
	d_AddVector(d_input, d_summands, d_output, sliceelements, slices);
}

__declspec(dllexport) void SubtractFromSlicesHalf(half* d_input, half* d_subtrahends, half* d_output, size_t sliceelements, uint slices)
{
	d_SubtractVector(d_input, d_subtrahends, d_output, sliceelements, slices);
}

__declspec(dllexport) void MultiplySlicesHalf(half* d_input, half* d_multiplicators, half* d_output, size_t sliceelements, uint slices)
{
	d_MultiplyByVector(d_input, d_multiplicators, d_output, sliceelements, slices);
}

__declspec(dllexport) void MultiplyComplexSlicesByScalar(float2* d_input, float* d_multiplicators, float2* d_output, size_t sliceelements, uint slices)
{
	d_ComplexMultiplyByVector(d_input, d_multiplicators, d_output, sliceelements, slices);
}

__declspec(dllexport) void DivideComplexSlicesByScalar(float2* d_input, float* d_multiplicators, float2* d_output, size_t sliceelements, uint slices)
{
	d_ComplexDivideByVector(d_input, d_multiplicators, d_output, sliceelements, slices);
}