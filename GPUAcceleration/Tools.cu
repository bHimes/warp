#include "hip/hip_runtime.h"
#include "Functions.h"
#include "../../gtom/include/CubicInterp.cuh"
using namespace gtom;

__declspec(dllexport) void FFT(float* d_input, float2* d_output, int3 dims, uint batch)
{
    d_FFTR2C(d_input, d_output, DimensionCount(dims), dims, batch);
}

__declspec(dllexport) void IFFT(float2* d_input, float* d_output, int3 dims, uint batch)
{
    d_IFFTC2R(d_input, d_output, DimensionCount(dims), dims, batch);
}

__declspec(dllexport) void Pad(float* d_input, float* d_output, int3 olddims, int3 newdims, uint batch)
{
    d_Pad(d_input, d_output, olddims, newdims, T_PAD_VALUE, 0.0f, batch);
}

__declspec(dllexport) void PadFT(float2* d_input, float2* d_output, int3 olddims, int3 newdims, uint batch)
{
    d_FFTPad(d_input, d_output, olddims, newdims, batch);
}

__declspec(dllexport) void CropFT(float2* d_input, float2* d_output, int3 olddims, int3 newdims, uint batch)
{
    d_FFTCrop(d_input, d_output, olddims, newdims, batch);
}

__declspec(dllexport) void RemapToFTComplex(float2* d_input, float2* d_output, int3 dims, uint batch)
{
    d_RemapHalfFFT2Half(d_input, d_output, dims, batch);
}

__declspec(dllexport) void RemapToFTFloat(float* d_input, float* d_output, int3 dims, uint batch)
{
    d_RemapHalfFFT2Half(d_input, d_output, dims, batch);
}

__declspec(dllexport) void RemapFromFTComplex(float2* d_input, float2* d_output, int3 dims, uint batch)
{
    d_RemapHalf2HalfFFT(d_input, d_output, dims, batch);
}

__declspec(dllexport) void RemapFromFTFloat(float* d_input, float* d_output, int3 dims, uint batch)
{
    d_RemapHalf2HalfFFT(d_input, d_output, dims, batch);
}

__declspec(dllexport) void RemapFullToFTFloat(float* d_input, float* d_output, int3 dims, uint batch)
{
    d_RemapFullFFT2Full(d_input, d_output, dims, batch);
}

__declspec(dllexport) void RemapFullFromFTFloat(float* d_input, float* d_output, int3 dims, uint batch)
{
    d_RemapFull2FullFFT(d_input, d_output, dims, batch);
}

__declspec(dllexport) void Extract(float* d_input, float* d_output, int3 dims, int3 dimsregion, int3* h_origins, uint batch)
{
	int3* d_origins = (int3*)CudaMallocFromHostArray(h_origins, batch * sizeof(int3));

	d_ExtractMany(d_input, d_output, dims, dimsregion, d_origins, batch);

	hipFree(d_origins);
}

__declspec(dllexport) void ExtractHalf(half* d_input, half* d_output, int3 dims, int3 dimsregion, int3* h_origins, uint batch)
{
	int3* d_origins = (int3*)CudaMallocFromHostArray(h_origins, batch * sizeof(int3));

	d_ExtractMany(d_input, d_output, dims, dimsregion, d_origins, batch);

	hipFree(d_origins);
}

__declspec(dllexport) void ReduceMean(float* d_input, float* d_output, uint vectorlength, uint nvectors, uint batch)
{
	d_ReduceMean(d_input, d_output, vectorlength, nvectors, batch);
}

__declspec(dllexport) void ReduceMeanHalf(half* d_input, half* d_output, uint vectorlength, uint nvectors, uint batch)
{
	d_ReduceMean(d_input, d_output, vectorlength, nvectors, batch);
}

__declspec(dllexport) void Normalize(float* d_ps, float* d_output, uint length, uint batch)
{
	d_NormMonolithic(d_ps, d_output, length, T_NORM_MEAN01STD, batch);
}

__declspec(dllexport) void NormalizeMasked(float* d_ps, float* d_output, float* d_mask, uint length, uint batch)
{
	d_NormMonolithic(d_ps, d_output, length, d_mask, T_NORM_MEAN01STD, batch);
}

__declspec(dllexport) void SphereMask(float* d_input, float* d_output, int3 dims, float radius, float sigma, uint batch)
{
	d_SphereMask(d_input, d_output, dims, &radius, sigma, NULL, batch);
}

__declspec(dllexport) void CreateCTF(float* d_output, float2* d_coords, uint length, CTFParams* h_params, bool amplitudesquared, uint batch)
{
	d_CTFSimulate(h_params, d_coords, d_output, length, amplitudesquared, false, batch);
}

__declspec(dllexport) void Resize(float* d_input, int3 dimsinput, float* d_output, int3 dimsoutput, uint batch)
{
	d_Scale(d_input, d_output, dimsinput, dimsoutput, T_INTERP_FOURIER, NULL, NULL, batch);
}

__declspec(dllexport) void ShiftStack(float* d_input, float* d_output, int3 dims, float* h_shifts, uint batch)
{
	d_Shift(d_input, d_output, dims, (tfloat3*)h_shifts, NULL, NULL, NULL, batch);
}

__declspec(dllexport) void ShiftStackMassive(float* d_input, float* d_output, int3 dims, float* h_shifts, uint batch)
{
	hipfftHandle planforw = d_FFTR2CGetPlan(DimensionCount(dims), dims);
	hipfftHandle planback = d_IFFTC2RGetPlan(DimensionCount(dims), dims);
	float2* d_intermediate;
	hipMalloc((void**)&d_intermediate, ElementsFFT(dims) * sizeof(float2));

	for (int b = 0; b < batch; b++)
		d_Shift(d_input + Elements(dims) * b, d_output + Elements(dims) * b, dims, (tfloat3*)h_shifts + b, &planforw, &planback, d_intermediate);

	hipfftDestroy(planforw);
	hipfftDestroy(planback);
	hipFree(d_intermediate);
}

__declspec(dllexport) void Cart2Polar(float* d_input, float* d_output, int2 dims, uint innerradius, uint exclusiveouterradius, uint batch)
{
	d_Cart2Polar(d_input, d_output, dims, T_INTERP_LINEAR, innerradius, exclusiveouterradius, batch);
}

__declspec(dllexport) void Cart2PolarFFT(float* d_input, float* d_output, int2 dims, uint innerradius, uint exclusiveouterradius, uint batch)
{
	d_Cart2PolarFFT(d_input, d_output, dims, T_INTERP_LINEAR, innerradius, exclusiveouterradius, batch);
}

__declspec(dllexport) void Xray(float* d_input, float* d_output, float ndevs, int2 dims, uint batch)
{
    d_Xray(d_input, d_output, toInt3(dims), ndevs, 5, batch);
}

// Arithmetics:

__declspec(dllexport) void Sum(float* d_input, float* d_output, uint length, uint batch)
{
    d_SumMonolithic(d_input, d_output, length, batch);
}

__declspec(dllexport) void Abs(float* d_input, float* d_output, size_t length)
{
    d_Abs(d_input, d_output, length);
}

__declspec(dllexport) void Amplitudes(float2* d_input, float* d_output, size_t length)
{
    d_Abs(d_input, d_output, length);
}

__declspec(dllexport) void Sign(float* d_input, float* d_output, size_t length)
{
    d_Sign(d_input, d_output, length);
}

__declspec(dllexport) void AddToSlices(float* d_input, float* d_summands, float* d_output, size_t sliceelements, uint slices)
{
	d_AddVector(d_input, d_summands, d_output, sliceelements, slices);
}

__declspec(dllexport) void SubtractFromSlices(float* d_input, float* d_subtrahends, float* d_output, size_t sliceelements, uint slices)
{
	d_SubtractVector(d_input, d_subtrahends, d_output, sliceelements, slices);
}

__declspec(dllexport) void MultiplySlices(float* d_input, float* d_multiplicators, float* d_output, size_t sliceelements, uint slices)
{
	d_MultiplyByVector(d_input, d_multiplicators, d_output, sliceelements, slices);
}

__declspec(dllexport) void DivideSlices(float* d_input, float* d_divisors, float* d_output, size_t sliceelements, uint slices)
{
	d_DivideSafeByVector(d_input, d_divisors, d_output, sliceelements, slices);
}

__declspec(dllexport) void AddToSlicesHalf(half* d_input, half* d_summands, half* d_output, size_t sliceelements, uint slices)
{
	d_AddVector(d_input, d_summands, d_output, sliceelements, slices);
}

__declspec(dllexport) void SubtractFromSlicesHalf(half* d_input, half* d_subtrahends, half* d_output, size_t sliceelements, uint slices)
{
	d_SubtractVector(d_input, d_subtrahends, d_output, sliceelements, slices);
}

__declspec(dllexport) void MultiplySlicesHalf(half* d_input, half* d_multiplicators, half* d_output, size_t sliceelements, uint slices)
{
	d_MultiplyByVector(d_input, d_multiplicators, d_output, sliceelements, slices);
}

__declspec(dllexport) void MultiplyComplexSlicesByScalar(float2* d_input, float* d_multiplicators, float2* d_output, size_t sliceelements, uint slices)
{
	d_ComplexMultiplyByVector(d_input, d_multiplicators, d_output, sliceelements, slices);
}

__declspec(dllexport) void DivideComplexSlicesByScalar(float2* d_input, float* d_multiplicators, float2* d_output, size_t sliceelements, uint slices)
{
	d_ComplexDivideSafeByVector(d_input, d_multiplicators, d_output, sliceelements, slices);
}

__declspec(dllexport) void Scale(float* d_input, float* d_output, int3 dimsinput, int3 dimsoutput, uint batch)
{
	d_Scale(d_input, d_output, dimsinput, dimsoutput, T_INTERP_FOURIER, NULL, NULL, batch);
}

__declspec(dllexport) void ProjectForward(float2* d_inputft, float2* d_outputft, int3 dimsinput, int2 dimsoutput, float3* h_angles, float supersample, uint batch)
{
    d_rlnProject(d_inputft, dimsinput, d_outputft, toInt3(dimsoutput), (tfloat3*)h_angles, supersample, batch);
}

__declspec(dllexport) void ProjectBackward(float2* d_volumeft, float* d_volumeweights, int3 dimsvolume, float2* d_projft, float* d_projweights, int2 dimsproj, int rmax, float3* h_angles, float supersample, uint batch)
{
	/*tfloat* d_amps = CudaMallocValueFilled(ElementsFFT(dimsvolume), (tfloat)0);
	d_Abs(d_projft, d_amps, ElementsFFT2(dimsproj));
	d_WriteMRC(d_amps, toInt3FFT(dimsproj), "d_amps.mrc");

	d_WriteMRC(d_projweights, toInt3(dimsproj.x / 2 + 1, dimsproj.y, batch), "d_projweights.mrc");

	tfloat* d_dummy = CudaMallocValueFilled(ElementsFFT2(dimsproj) * batch * 2, 1.0f);*/

    d_rlnBackproject(d_volumeft, d_volumeweights, dimsvolume, d_projft, d_projweights, toInt3(dimsproj), rmax, (tfloat3*)h_angles, supersample, batch);

	/*d_Abs(d_volumeft, d_amps, ElementsFFT(dimsvolume));
	d_WriteMRC(d_amps, toInt3FFT(dimsvolume), "d_volamps.mrc");*/
}

__declspec(dllexport) void Bandpass(float* d_input, float* d_output, int3 dims, float nyquistlow, float nyquisthigh, uint batch)
{
    d_BandpassNonCubic(d_input, d_output, dims, nyquistlow, nyquisthigh, batch);
}

__declspec(dllexport) void Rotate2D(float* d_input, float* d_output, int2 dims, float* h_angles, int oversample, uint batch)
{
	if (oversample <= 1)
	{
		d_Rotate2D(d_input, d_output, dims, h_angles, T_INTERP_CUBIC, true, batch);
	}
	else
	{
		int2 dimspadded = dims * oversample;
	    float* d_temp;
		hipMalloc((void**)&d_temp, Elements2(dimspadded) * sizeof(float));

		for (int b = 0; b < batch; b++)
		{
		    d_Scale(d_input + Elements2(dims) * b, d_temp, toInt3(dims), toInt3(dimspadded), T_INTERP_FOURIER);
			d_Rotate2D(d_temp, d_temp, dimspadded, h_angles + b, T_INTERP_CUBIC, true, 1);
			d_Scale(d_temp, d_output + Elements2(dims) * b, toInt3(dimspadded), toInt3(dims), T_INTERP_FOURIER);
		}

		hipFree(d_temp);
	}
}

__global__ void ShiftAndRotate2DKernel(float* d_input, float* d_output, int2 dims, int2 dimsori, glm::mat3* d_transforms)
{
	uint idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= dims.x)
		return;
	uint idy = blockIdx.y * blockDim.y + threadIdx.y;
	if (idy >= dims.y)
		return;

	d_input += Elements2(dimsori) * blockIdx.z;

	int x, y;
	x = idx;
	y = idy;

	glm::vec3 pos = d_transforms[blockIdx.z] * glm::vec3(x - dims.x / 2, y - dims.y / 2, 1.0f) + glm::vec3(dimsori.x / 2, dimsori.y / 2, 0.0f);
	
	float val = 0;
	if (pos.x >= 0 && pos.x < dims.x && pos.y >= 0 && pos.y < dims.y)
	{
	    int x0 = floor(pos.x);
		int x1 = tmin(x0 + 1, dims.x - 1);
		pos.x -= x0;

		int y0 = floor(pos.y);
		int y1 = tmin(y0 + 1, dims.y - 1);
		pos.y -= y0;

		float d000 = d_input[y0 * dimsori.x + x0];
		float d001 = d_input[y0 * dimsori.x + x1];
		float d010 = d_input[y1 * dimsori.x + x0];
		float d011 = d_input[y1 * dimsori.x + x1];

		float dx00 = lerp(d000, d001, pos.x);
		float dx01 = lerp(d010, d011, pos.x);

		val = lerp(dx00, dx01, pos.y);
	}

	d_output[(blockIdx.z * dims.y + idy) * dims.x + idx] = val;
}

__declspec(dllexport) void ShiftAndRotate2D(float* d_input, float* d_output, int2 dims, float2* h_shifts, float* h_angles, uint batch)
{
	glm::mat3* h_transforms = (glm::mat3*)malloc(batch * sizeof(glm::mat3));
	for (uint b = 0; b < batch; b++)
		h_transforms[b] = Matrix3RotationZ(-h_angles[b]) * Matrix3Translation(tfloat2(-h_shifts[b].x, -h_shifts[b].y));
	glm::mat3* d_transforms = (glm::mat3*)CudaMallocFromHostArray(h_transforms, batch * sizeof(glm::mat3));
	free(h_transforms);

	dim3 TpB = dim3(16, 16);
	dim3 grid = dim3((dims.x + 15) / 16, (dims.y + 15) / 16, batch);

	ShiftAndRotate2DKernel << <grid, TpB >> > (d_input, d_output, dims, dims * 1, d_transforms);

	hipFree(d_transforms);
}

__declspec(dllexport) int CreateFFTPlan(int3 dims, uint batch)
{
    return d_FFTR2CGetPlan(DimensionCount(dims), dims, batch);
}

__declspec(dllexport) int CreateIFFTPlan(int3 dims, uint batch)
{
    return d_IFFTC2RGetPlan(DimensionCount(dims), dims, batch);
}

__declspec(dllexport) void DestroyFFTPlan(hipfftHandle plan)
{
    hipfftDestroy(plan);
}