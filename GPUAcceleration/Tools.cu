#include "hip/hip_runtime.h"
#include "Functions.h"
using namespace gtom;


__declspec(dllexport) void Extract(float* d_input, float* d_output, int3 dims, int3 dimsregion, int3* h_origins, uint batch)
{
	int3* d_origins = (int3*)CudaMallocFromHostArray(h_origins, batch * sizeof(int3));

	d_ExtractMany(d_input, d_output, dims, dimsregion, d_origins, batch);

	hipFree(d_origins);
}

__declspec(dllexport) void ExtractHalf(half* d_input, half* d_output, int3 dims, int3 dimsregion, int3* h_origins, uint batch)
{
	int3* d_origins = (int3*)CudaMallocFromHostArray(h_origins, batch * sizeof(int3));

	d_ExtractMany(d_input, d_output, dims, dimsregion, d_origins, batch);

	hipFree(d_origins);
}

__declspec(dllexport) void ReduceMean(float* d_input, float* d_output, uint vectorlength, uint nvectors, uint batch)
{
	d_ReduceMean(d_input, d_output, vectorlength, nvectors, batch);
}

__declspec(dllexport) void ReduceMeanHalf(half* d_input, half* d_output, uint vectorlength, uint nvectors, uint batch)
{
	d_ReduceMean(d_input, d_output, vectorlength, nvectors, batch);
}

__declspec(dllexport) void CreateCTF(float* d_output, float2* d_coords, uint length, CTFParams* h_params, bool amplitudesquared, uint batch)
{
	d_CTFSimulate(h_params, d_coords, d_output, length, amplitudesquared, batch);
}

__declspec(dllexport) void Resize(float* d_input, int3 dimsinput, float* d_output, int3 dimsoutput, uint batch)
{
	d_Scale(d_input, d_output, dimsinput, dimsoutput, T_INTERP_FOURIER, NULL, NULL, batch);
}

__declspec(dllexport) void ShiftStack(float* d_input, float* d_output, int3 dims, float3* h_shifts, uint batch)
{
	d_Shift(d_input, d_output, dims, (tfloat3*)h_shifts, NULL, NULL, NULL, batch);
}

__declspec(dllexport) void Add(float* d_input, float* d_summands, float* d_output, size_t elements)
{
	d_AddVector(d_input, d_summands, d_output, elements);
}

__declspec(dllexport) void AddToSlices(float* d_input, float* d_summands, float* d_output, size_t sliceelements, uint slices)
{
	d_AddVector(d_input, d_summands, d_output, sliceelements, slices);
}

__declspec(dllexport) void Subtract(float* d_input, float* d_subtrahends, float* d_output, size_t elements)
{
	d_SubtractVector(d_input, d_subtrahends, d_output, elements);
}

__declspec(dllexport) void SubtractFromSlices(float* d_input, float* d_subtrahends, float* d_output, size_t sliceelements, uint slices)
{
	d_SubtractVector(d_input, d_subtrahends, d_output, sliceelements, slices);
}

__declspec(dllexport) void Multiply(float* d_input, float* d_multiplicators, float* d_output, size_t elements)
{
	d_MultiplyByVector(d_input, d_multiplicators, d_output, elements);
}

__declspec(dllexport) void MultiplySlices(float* d_input, float* d_multiplicators, float* d_output, size_t sliceelements, uint slices)
{
	d_MultiplyByVector(d_input, d_multiplicators, d_output, sliceelements, slices);
}

__declspec(dllexport) void Divide(float* d_input, float* d_divisors, float* d_output, size_t elements)
{
	d_MultiplyByVector(d_input, d_divisors, d_output, elements);
}

__declspec(dllexport) void DivideSlices(float* d_input, float* d_divisors, float* d_output, size_t sliceelements, uint slices)
{
	d_MultiplyByVector(d_input, d_divisors, d_output, sliceelements, slices);
}

__declspec(dllexport) void AddHalf(half* d_input, half* d_summands, half* d_output, size_t elements)
{
	d_AddVector(d_input, d_summands, d_output, elements);
}

__declspec(dllexport) void AddToSlicesHalf(half* d_input, half* d_summands, half* d_output, size_t sliceelements, uint slices)
{
	d_AddVector(d_input, d_summands, d_output, sliceelements, slices);
}

__declspec(dllexport) void SubtractHalf(half* d_input, half* d_subtrahends, half* d_output, size_t elements)
{
	d_SubtractVector(d_input, d_subtrahends, d_output, elements);
}

__declspec(dllexport) void SubtractFromSlicesHalf(half* d_input, half* d_subtrahends, half* d_output, size_t sliceelements, uint slices)
{
	d_SubtractVector(d_input, d_subtrahends, d_output, sliceelements, slices);
}

__declspec(dllexport) void MultiplyHalf(half* d_input, half* d_multiplicators, half* d_output, size_t elements)
{
	d_MultiplyByVector(d_input, d_multiplicators, d_output, elements);
}

__declspec(dllexport) void MultiplySlicesHalf(half* d_input, half* d_multiplicators, half* d_output, size_t sliceelements, uint slices)
{
	d_MultiplyByVector(d_input, d_multiplicators, d_output, sliceelements, slices);
}