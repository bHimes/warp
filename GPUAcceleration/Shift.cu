#include "hip/hip_runtime.h"
#include "Functions.h"
#include <hip/device_functions.h>
using namespace gtom;

#define SHIFT_THREADS 128

__global__ void ShiftGetAverageKernel(float2* d_phase, float2* d_average, float2* d_shiftfactors, float2* d_shifts, uint length, uint probelength, uint nspectra, uint nframes);
__global__ void ShiftGetDiffKernel(float2* d_phase, float2* d_average, float2* d_shiftfactors, uint length, uint probelength, float2* d_shifts, float* d_diff);
__global__ void ShiftGetGradKernel(float2* d_phase, float2* d_average, float2* d_shiftfactors, uint length, uint probelength, float2* d_shifts, float2* d_grad);

/*

Supplied with a stack of frames, extraction positions for sub-regions, and a mask of relevant pixels in Fspace, 
this method extracts portions of each frame, computes the FT, and returns the relevant pixels.

*/

__declspec(dllexport) void CreateShift(float* d_frame,
										int2 dimsframe,
										int nframes,
										int3* h_origins,
										int norigins,
										int2 dimsregion,
										size_t* h_mask,
										uint masklength,
										float2* d_outputall)
{
	int2 dimsunpadded = toInt2(dimsregion.x / 1, dimsregion.y / 1);

	int3* d_origins = (int3*)CudaMallocFromHostArray(h_origins, norigins * sizeof(int3));
	size_t* d_mask = (size_t*)CudaMallocFromHostArray(h_mask, masklength * sizeof(size_t));
	tfloat* d_temp;
	hipMalloc((void**)&d_temp, norigins * ElementsFFT2(dimsregion) * sizeof(tcomplex));
	tcomplex* d_tempft;
	hipMalloc((void**)&d_tempft, norigins * ElementsFFT2(dimsregion) * sizeof(tcomplex));
	tcomplex* d_dense;
	hipMalloc((void**)&d_dense, norigins * masklength * sizeof(tcomplex));

	for (uint z = 0; z < nframes; z++)
	{
		d_ExtractMany(d_frame + Elements2(dimsframe) * z, d_temp, toInt3(dimsframe), toInt3(dimsregion), d_origins, norigins);
		d_NormMonolithic(d_temp, d_temp, Elements2(dimsregion), T_NORM_MEAN01STD, norigins);
		d_HammingMask(d_temp, d_temp, toInt3(dimsregion), NULL, NULL, norigins);
		//d_WriteMRC(d_temp, toInt3(dimsregion.x, dimsregion.y, norigins), "d_shifttemp.mrc");
		d_FFTR2C(d_temp, d_tempft, 2, toInt3(dimsregion), norigins);
		d_RemapHalfFFT2Half(d_tempft, (tcomplex*)d_temp, toInt3(dimsregion), norigins);
		d_Remap((tcomplex*)d_temp, d_mask, d_outputall + masklength * norigins * z, masklength, ElementsFFT2(dimsregion), make_hipComplex(0.0f, 0.0f), norigins);
		//d_ComplexNormalize(d_dense, d_dense, masklength * norigins);
	}

	hipFree(d_dense);
	hipFree(d_tempft);
	hipFree(d_temp);
	hipFree(d_mask);
	hipFree(d_origins);
}

__declspec(dllexport) void ShiftGetAverage(float2* d_phase, 
											float2* d_average, 
											float2* d_shiftfactors,
											uint length,  
											uint probelength,
											float2* d_shifts, 
											uint npositions, 
											uint nframes)
{
	float2* d_shiftshalf;
	hipMalloc((void**)&d_shiftshalf, npositions * nframes * sizeof(float2));
	d_ConvertTFloatTo((float*)d_shifts, (float*)d_shiftshalf, npositions * nframes * 2);
	
	int TpB = tmin(SHIFT_THREADS, NextMultipleOf(length, 32));
	dim3 grid = dim3((length + TpB - 1) / TpB, npositions, 1);
	ShiftGetAverageKernel <<<grid, TpB>>> (d_phase, d_average, d_shiftfactors, d_shiftshalf, length, probelength, npositions, nframes);

	/*float2* d_averagef;
	hipMalloc((void**)&d_averagef, length * npositions * sizeof(float2));
	d_ConvertToTFloat((half*)d_average, (float*)d_averagef, npositions * length * 2);
	float2* h_averagef = (float2*)MallocFromDeviceArray(d_averagef, length * npositions * sizeof(float2));
	hipFree(d_averagef);
	
	float2* d_phasef;
	hipMalloc((void**)&d_phasef, length * npositions * nframes * sizeof(float2));
	d_ConvertToTFloat((half*)d_phase, (float*)d_phasef, npositions * nframes * length * 2);
	float2* h_phasef = (float2*)MallocFromDeviceArray(d_phasef, length * npositions * nframes * sizeof(float2));
	hipFree(d_phasef);

	free(h_averagef);
	free(h_phasef);*/

	hipFree(d_shiftshalf);
}

__global__ void ShiftGetAverageKernel(float2* d_phase, float2* d_average, float2* d_shiftfactors, float2* d_shifts, uint length, uint probelength, uint npositions, uint nframes)
{
	d_phase += blockIdx.y * length;
	d_average += blockIdx.y * probelength;
	d_shifts += blockIdx.y;

	__shared__ float2 s_shifts[256];	// 256 frames should be enough for everyone
	for (uint i = threadIdx.x; i < nframes; i += blockDim.x)
		s_shifts[i] = d_shifts[npositions * i];
	__syncthreads();


	for (uint id = blockIdx.x * blockDim.x + threadIdx.x; 
		 id < probelength; 
		 id += gridDim.x * blockDim.x)
	{
		float2 shiftfactors = d_shiftfactors[id];
		float2 sum = make_float2(0.0f, 0.0f);

		for (uint frame = 0; frame < nframes; frame++)
		{
			float2 shift = s_shifts[frame];
			float phase = shiftfactors.x * shift.x + shiftfactors.y * shift.y;
			float2 change = make_float2(__cosf(phase), __sinf(phase));

			float2 value = d_phase[length * npositions * frame + id];
			value = hipCmulf(value, change);

			sum += value;
		}
		
		sum /= nframes;

		d_average[id] = sum;
	}
}

__declspec(dllexport) void ShiftGetDiff(float2* d_phase, 
											float2* d_average, 
											float2* d_shiftfactors, 
											uint length, 
											uint probelength,
											float2* d_shifts,
											float* h_diff, 
											uint npositions, 
											uint nframes)
{
	int TpB = tmin(SHIFT_THREADS, NextMultipleOf(probelength, 32));
	dim3 grid = dim3(npositions, nframes, 1);

	float* d_diff;
	hipMalloc((void**)&d_diff, npositions * nframes * grid.x * sizeof(float));
	float* d_diffreduced;
	hipMalloc((void**)&d_diffreduced, npositions * nframes * sizeof(float));

	ShiftGetDiffKernel <<<grid, TpB>>> (d_phase, d_average, d_shiftfactors, length, probelength, d_shifts, d_diff);

	//d_SumMonolithic(d_diff, d_diffreduced, grid.x, npositions * nframes);
	hipMemcpy(h_diff, d_diff, npositions * nframes * sizeof(float), hipMemcpyDeviceToHost);
	
	hipFree(d_diffreduced);
	hipFree(d_diff);
}

__global__ void ShiftGetDiffKernel(float2* d_phase, float2* d_average, float2* d_shiftfactors, uint length, uint probelength, float2* d_shifts, float* d_diff)
{
	__shared__ float s_diff[SHIFT_THREADS];
	s_diff[threadIdx.x] = 0.0f;
	__shared__ float s_ampsum[SHIFT_THREADS];
	s_ampsum[threadIdx.x] = 0.0f;

	uint specid = blockIdx.y * gridDim.x + blockIdx.x;
	d_phase += specid * length;
	d_average += blockIdx.x * probelength;

	float2 shift = d_shifts[specid];
	float diffsum = 0.0f;
	float ampsum = 0.0f;

	for (uint id = threadIdx.x; id < probelength; id += blockDim.x)
	{
		float2 value = d_phase[id];
		float2 average = d_average[id];

		float2 shiftfactors = d_shiftfactors[id];

		float phase = shiftfactors.x * shift.x + shiftfactors.y * shift.y;
		float2 change = make_float2(__cosf(phase), __sinf(phase));

		value = hipCmulf(value, change);

		float2 valuenorm = value / tmax(1e-10f, sqrt(value.x * value.x + value.y * value.y));
		float avgamp = tmax(1e-10f, sqrt(average.x * average.x + average.y * average.y));
		average /= avgamp;

		float diff = acos(tmax(-1.0f, tmin(valuenorm.x * average.x + valuenorm.y * average.y, 1.0f))) * avgamp;
		//float diff = dotp2(value, average);
		diffsum += diff;
		ampsum += avgamp;
	}

	s_diff[threadIdx.x] = diffsum;
	s_ampsum[threadIdx.x] = ampsum;
	__syncthreads();

	if (threadIdx.x == 0)
	{
		for (uint id = 1; id < blockDim.x; id++)
		{
			diffsum += s_diff[id];
			ampsum += s_ampsum[id];
		}

		d_diff[specid] = diffsum / ampsum;
	}
}

__declspec(dllexport) void ShiftGetGrad(float2* d_phase, 
										float2* d_average, 
										float2* d_shiftfactors, 
										uint length, 
										uint probelength,
										float2* d_shifts,
										float2* h_grad, 
										uint npositions, 
										uint nframes)
{
	int TpB = tmin(SHIFT_THREADS, NextMultipleOf(probelength, 32));
	dim3 grid = dim3(npositions, nframes, 1);

	float2* d_grad;
	hipMalloc((void**)&d_grad, npositions * nframes * grid.x * sizeof(float2));
	float2* d_gradreduced;
	hipMalloc((void**)&d_gradreduced, npositions * nframes * sizeof(float2));

	ShiftGetGradKernel <<<grid, TpB>>> (d_phase, d_average, d_shiftfactors, length, probelength, d_shifts, d_grad);

	float2* h_grad2 = (float2*)MallocFromDeviceArray(d_grad, npositions * nframes * grid.x * sizeof(float2));
	free(h_grad2);

	//d_SumMonolithic(d_grad, d_gradreduced, grid.x, npositions * nframes);
	hipMemcpy(h_grad, d_grad, npositions * nframes * sizeof(float2), hipMemcpyDeviceToHost);
	
	hipFree(d_gradreduced);
	hipFree(d_grad);
}

__global__ void ShiftGetGradKernel(float2* d_phase, 
									float2* d_average, 
									float2* d_shiftfactors, 
									uint length, 
									uint probelength, 
									float2* d_shifts, 
									float2* d_grad)
{
	__shared__ float2 s_grad[SHIFT_THREADS];
	s_grad[threadIdx.x] = make_float2(0.0f, 0.0f);
	__shared__ float s_ampsum[SHIFT_THREADS];
	s_ampsum[threadIdx.x] = 0.0f;

	uint specid = blockIdx.y * gridDim.x + blockIdx.x;
	d_phase += specid * length;
	d_average += blockIdx.x * probelength;

	float2 shift = d_shifts[specid];
	float2 gradsum = make_float2(0.0f, 0.0f);
	float ampsum = 0.0f;

	for (uint id = threadIdx.x; id < probelength; id += blockDim.x)
	{
		float2 value = d_phase[id];
		float2 average = d_average[id];

		float2 shiftfactors = d_shiftfactors[id];
		float weight = tmax(1e-10f, sqrt(average.x * average.x + average.y * average.y));// __half2float(d_weights[id]);

		float phase = shiftfactors.x * shift.x + shiftfactors.y * shift.y;
		float2 change = make_float2(__cosf(phase), __sinf(phase));
		float2 altvalue = cmul(value, change);
		
		gradsum.x += -sgn(altvalue.x * average.y - altvalue.y * average.x) * shiftfactors.x * weight;
		gradsum.y += -sgn(altvalue.x * average.y - altvalue.y * average.x) * shiftfactors.y * weight;
		ampsum += weight;
	}

	s_grad[threadIdx.x] = gradsum;
	s_ampsum[threadIdx.x] = ampsum;
	__syncthreads();

	if (threadIdx.x == 0)
	{
		for (uint id = 1; id < blockDim.x; id++)
		{
			gradsum = gradsum + s_grad[id];
			ampsum = ampsum + s_ampsum[id];
		}

		d_grad[specid] = gradsum / ampsum;
	}
}

__declspec(dllexport) void CreateMotionBlur(float* d_output, int3 dims, float* h_shifts, uint nshifts, uint batch)
{
    d_MotionBlur(d_output, dims, (float3*)h_shifts, nshifts, false, batch);
}