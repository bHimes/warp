#include "hip/hip_runtime.h"
#include "Functions.h"
#include <hip/device_functions.h>
using namespace gtom;

#define SHIFT_THREADS 128

__global__ void ShiftGetAverageKernel(half2* d_phase, half2* d_average, half2* d_shiftfactors, half2* d_shifts, uint length, uint probelength, uint nspectra, uint nframes);
__global__ void ShiftGetDiffKernel(half2* d_phase, half2* d_average, half2* d_shiftfactors, half* d_weights, uint length, uint probelength, half2* d_shifts, float* d_diff);
__global__ void ShiftGetGradKernel(half2* d_phase, half2* d_average, half2* d_shiftfactors, half* d_weights, uint length, uint probelength, half2* d_shifts, float2* d_grad);

/*

Supplied with a stack of frames, extraction positions for sub-regions, and a mask of relevant pixels in Fspace, 
this method extracts portions of each frame, computes the FT, and returns the relevant pixels.

*/

__declspec(dllexport) void CreateShift(float* d_frame,
										int2 dimsframe,
										int nframes,
										int3* h_origins,
										int norigins,
										int2 dimsregion,
										size_t* h_mask,
										uint masklength,
										half2* d_outputall)
{
	int2 dimsunpadded = toInt2(dimsregion.x / 1, dimsregion.y / 1);

	int3* d_origins = (int3*)CudaMallocFromHostArray(h_origins, norigins * sizeof(int3));
	size_t* d_mask = (size_t*)CudaMallocFromHostArray(h_mask, masklength * sizeof(size_t));
	tfloat* d_temp;
	hipMalloc((void**)&d_temp, norigins * ElementsFFT2(dimsregion) * sizeof(tcomplex));
	tcomplex* d_tempft;
	hipMalloc((void**)&d_tempft, norigins * ElementsFFT2(dimsregion) * sizeof(tcomplex));
	tcomplex* d_dense;
	hipMalloc((void**)&d_dense, norigins * masklength * sizeof(tcomplex));

	for (uint z = 0; z < nframes; z++)
	{
		d_ExtractMany(d_frame + Elements2(dimsframe) * z, d_temp, toInt3(dimsframe), toInt3(dimsregion), d_origins, norigins);
		d_NormMonolithic(d_temp, d_temp, Elements2(dimsregion), T_NORM_MEAN01STD, norigins);
		d_HammingMask(d_temp, d_temp, toInt3(dimsregion), NULL, NULL, norigins);
		//d_WriteMRC(d_temp, toInt3(dimsregion.x, dimsregion.y, norigins), "d_shifttemp.mrc");
		d_FFTR2C(d_temp, d_tempft, 2, toInt3(dimsregion), norigins);
		d_RemapHalfFFT2Half(d_tempft, (tcomplex*)d_temp, toInt3(dimsregion), norigins);
		d_Remap((tcomplex*)d_temp, d_mask, d_dense, masklength, ElementsFFT2(dimsregion), make_hipComplex(0.0f, 0.0f), norigins);
		d_ComplexNormalize(d_dense, d_dense, masklength * norigins);
		d_ConvertTFloatTo((tfloat*)d_dense, (half*)(d_outputall + masklength * norigins * z), masklength * norigins * 2);
	}

	hipFree(d_dense);
	hipFree(d_tempft);
	hipFree(d_temp);
	hipFree(d_mask);
	hipFree(d_origins);
}

__declspec(dllexport) void ShiftGetAverage(half2* d_phase, 
											half2* d_average, 
											half2* d_shiftfactors,
											uint length,  
											uint probelength,
											float2* d_shifts, 
											uint npositions, 
											uint nframes)
{
	half2* d_shiftshalf;
	hipMalloc((void**)&d_shiftshalf, npositions * nframes * sizeof(half2));
	d_ConvertTFloatTo((float*)d_shifts, (half*)d_shiftshalf, npositions * nframes * 2);
	
	int TpB = tmin(SHIFT_THREADS, NextMultipleOf(length, 32));
	dim3 grid = dim3((length + TpB - 1) / TpB, npositions, 1);
	ShiftGetAverageKernel <<<grid, TpB>>> (d_phase, d_average, d_shiftfactors, d_shiftshalf, length, probelength, npositions, nframes);

	/*float2* d_averagef;
	hipMalloc((void**)&d_averagef, length * npositions * sizeof(float2));
	d_ConvertToTFloat((half*)d_average, (float*)d_averagef, npositions * length * 2);
	float2* h_averagef = (float2*)MallocFromDeviceArray(d_averagef, length * npositions * sizeof(float2));
	hipFree(d_averagef);
	
	float2* d_phasef;
	hipMalloc((void**)&d_phasef, length * npositions * nframes * sizeof(float2));
	d_ConvertToTFloat((half*)d_phase, (float*)d_phasef, npositions * nframes * length * 2);
	float2* h_phasef = (float2*)MallocFromDeviceArray(d_phasef, length * npositions * nframes * sizeof(float2));
	hipFree(d_phasef);

	free(h_averagef);
	free(h_phasef);*/

	hipFree(d_shiftshalf);
}

__global__ void ShiftGetAverageKernel(half2* d_phase, half2* d_average, half2* d_shiftfactors, half2* d_shifts, uint length, uint probelength, uint npositions, uint nframes)
{
	d_phase += blockIdx.y * length;
	d_average += blockIdx.y * probelength;
	d_shifts += blockIdx.y;

	__shared__ half2 s_shifts[256];	// 256 frames should be enough for everyone
	for (uint i = threadIdx.x; i < nframes; i += blockDim.x)
		s_shifts[i] = d_shifts[npositions * i];
	__syncthreads();


	for (uint id = blockIdx.x * blockDim.x + threadIdx.x; 
		 id < probelength; 
		 id += gridDim.x * blockDim.x)
	{
		float2 shiftfactors = __half22float2(d_shiftfactors[id]);
		float2 sum = make_float2(0.0f, 0.0f);

		for (uint frame = 0; frame < nframes; frame++)
		{
			float2 shift = __half22float2(s_shifts[frame]);
			float phase = shiftfactors.x * shift.x + shiftfactors.y * shift.y;
			float2 change = make_float2(__cosf(phase), __sinf(phase));

			float2 value = __half22float2(d_phase[length * npositions * frame + id]);
			value = hipCmulf(value, change);

			sum += value;
		}
		
		float normalization = 1.0f / nframes;
		sum = make_float2(sum.x * normalization, sum.y * normalization);

		d_average[id] = __float22half2_rn(sum);
	}
}

__declspec(dllexport) void ShiftGetDiff(half2* d_phase, 
											half2* d_average, 
											half2* d_shiftfactors, 
											half* d_weights,
											uint length, 
											uint probelength,
											float2* d_shifts,
											float* h_diff, 
											uint npositions, 
											uint nframes)
{
	half2* d_shiftshalf;
	hipMalloc((void**)&d_shiftshalf, npositions * nframes * sizeof(half2));
	d_ConvertTFloatTo((float*)d_shifts, (half*)d_shiftshalf, npositions * nframes * 2);

	int TpB = tmin(SHIFT_THREADS, NextMultipleOf(probelength, 32));
	dim3 grid = dim3(tmin(128, (probelength + TpB - 1) / TpB), npositions, nframes);

	float* d_diff;
	hipMalloc((void**)&d_diff, npositions * nframes * grid.x * sizeof(float));
	float* d_diffreduced;
	hipMalloc((void**)&d_diffreduced, npositions * nframes * sizeof(float));

	ShiftGetDiffKernel <<<grid, TpB>>> (d_phase, d_average, d_shiftfactors, d_weights, length, probelength, d_shiftshalf, d_diff);

	d_SumMonolithic(d_diff, d_diffreduced, grid.x, npositions * nframes);
	hipMemcpy(h_diff, d_diffreduced, npositions * nframes * sizeof(float), hipMemcpyDeviceToHost);
	
	hipFree(d_diffreduced);
	hipFree(d_diff);
	hipFree(d_shiftshalf);
}

__global__ void ShiftGetDiffKernel(half2* d_phase, half2* d_average, half2* d_shiftfactors, half* d_weights, uint length, uint probelength, half2* d_shifts, float* d_diff)
{
	__shared__ float s_diff[SHIFT_THREADS];
	s_diff[threadIdx.x] = 0.0f;

	uint specid = blockIdx.z * gridDim.y + blockIdx.y;
	d_phase += specid * length;
	d_average += blockIdx.y * probelength;
	d_weights += blockIdx.y * length;

	float2 shift = __half22float2(d_shifts[specid]);
	float diffsum = 0.0f;

	for (uint id = blockIdx.x * blockDim.x + threadIdx.x; 
		 id < probelength; 
		 id += gridDim.x * blockDim.x)
	{
		float2 value = __half22float2(d_phase[id]);
		float2 average = __half22float2(d_average[id]);

		float2 shiftfactors = __half22float2(d_shiftfactors[id]);

		float phase = shiftfactors.x * shift.x + shiftfactors.y * shift.y;
		float2 change = make_float2(__cosf(phase), __sinf(phase));

		value = hipCmulf(value, change);

		float diff = acos(tmax(-1.0f, tmin(value.x * average.x + value.y * average.y, 1.0f))) * __half2float(d_weights[id]);
		diffsum += diff;
	}

	s_diff[threadIdx.x] = diffsum;
	__syncthreads();

	if (threadIdx.x == 0)
	{
		for (uint id = 1; id < blockDim.x; id++)
			diffsum += s_diff[id];

		d_diff[specid * gridDim.x + blockIdx.x] = diffsum;
	}
}

__declspec(dllexport) void ShiftGetGrad(half2* d_phase, 
										half2* d_average, 
										half2* d_shiftfactors, 
										half* d_weights,
										uint length, 
										uint probelength,
										float2* d_shifts,
										float2* h_grad, 
										uint npositions, 
										uint nframes)
{
	half2* d_shiftshalf;
	hipMalloc((void**)&d_shiftshalf, npositions * nframes * sizeof(half2));
	d_ConvertTFloatTo((float*)d_shifts, (half*)d_shiftshalf, npositions * nframes * 2);

	int TpB = tmin(SHIFT_THREADS, NextMultipleOf(probelength, 32));
	dim3 grid = dim3(tmin(128, (probelength + TpB - 1) / TpB), npositions, nframes);

	float2* d_grad;
	hipMalloc((void**)&d_grad, npositions * nframes * grid.x * sizeof(float2));
	float2* d_gradreduced;
	hipMalloc((void**)&d_gradreduced, npositions * nframes * sizeof(float2));

	ShiftGetGradKernel <<<grid, TpB>>> (d_phase, d_average, d_shiftfactors, d_weights, length, probelength, d_shiftshalf, d_grad);

	float2* h_grad2 = (float2*)MallocFromDeviceArray(d_grad, npositions * nframes * grid.x * sizeof(float2));
	free(h_grad2);

	d_SumMonolithic(d_grad, d_gradreduced, grid.x, npositions * nframes);
	hipMemcpy(h_grad, d_gradreduced, npositions * nframes * sizeof(float2), hipMemcpyDeviceToHost);
	
	hipFree(d_gradreduced);
	hipFree(d_grad);
	hipFree(d_shiftshalf);
}

__global__ void ShiftGetGradKernel(half2* d_phase, half2* d_average, half2* d_shiftfactors, half* d_weights, uint length, uint probelength, half2* d_shifts, float2* d_grad)
{
	__shared__ float2 s_grad[SHIFT_THREADS];
	s_grad[threadIdx.x] = make_float2(0.0f, 0.0f);

	uint specid = blockIdx.z * gridDim.y + blockIdx.y;
	d_phase += specid * length;
	d_average += blockIdx.y * probelength;
	d_weights += blockIdx.y * length;

	float2 shift = __half22float2(d_shifts[specid]);
	float2 gradsum = make_float2(0.0f, 0.0f);

	for (uint id = blockIdx.x * blockDim.x + threadIdx.x; 
		 id < probelength; 
		 id += gridDim.x * blockDim.x)
	{
		float2 value = __half22float2(d_phase[id]);
		float2 average = __half22float2(d_average[id]);

		float2 shiftfactors = __half22float2(d_shiftfactors[id]);
		float weight = __half2float(d_weights[id]);

		float phase = shiftfactors.x * shift.x + shiftfactors.y * shift.y;
		float2 change = make_float2(__cosf(phase), __sinf(phase));
		float2 altvalue = cmul(value, change);
		
		gradsum.x += -sgn(altvalue.x * average.y - altvalue.y * average.x) * shiftfactors.x * weight;
		gradsum.y += -sgn(altvalue.x * average.y - altvalue.y * average.x) * shiftfactors.y * weight;
	}

	s_grad[threadIdx.x] = gradsum;
	__syncthreads();

	if (threadIdx.x == 0)
	{
		for (uint id = 1; id < blockDim.x; id++)
			gradsum = gradsum + s_grad[id];

		d_grad[specid * gridDim.x + blockIdx.x] = gradsum;
	}
}