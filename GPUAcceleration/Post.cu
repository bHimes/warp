#include "hip/hip_runtime.h"
#include "Functions.h"
using namespace gtom;

__declspec(dllexport) void GetMotionFilter(float* d_output, int3 dims, float3* h_shifts, uint nshifts, uint batch)
{
	tcomplex* d_phases = CudaMallocValueFilled(ElementsFFT2(dims) * nshifts * batch, make_hipComplex(1.0f, 0.0f));
	tcomplex* d_meanphases;
	hipMalloc((void**)&d_meanphases, ElementsFFT2(dims) * batch * sizeof(tcomplex));
	
	d_Shift(d_phases, d_phases, dims, (tfloat3*)h_shifts, true, nshifts * batch);
	d_ReduceMean(d_phases, d_meanphases, ElementsFFT2(dims), nshifts, batch);
	d_Abs(d_meanphases, d_output, ElementsFFT2(dims) * batch);

	hipFree(d_meanphases);
	hipFree(d_phases);
}

__declspec(dllexport) void WeightedFrameSum(float* d_frames, 
											float* d_ctf, 
											float* d_dose, 
											float* d_outputframes, 
											float* d_outputspectrum, 
											int2 dims, 
											uint nframes, 
											uint batch)
{
	float* d_framespectra;
	hipMalloc((void**)&d_framespectra, ElementsFFT2(dims) * nframes * batch * sizeof(float));
	d_MultiplyByVector(d_ctf, d_dose, d_framespectra, ElementsFFT2(dims) * nframes * batch);
	float* d_sumspectra;
	hipMalloc((void**)&d_sumspectra, ElementsFFT2(dims) * batch * sizeof(float));
	d_ReduceAdd(d_framespectra, d_sumspectra, ElementsFFT2(dims), nframes, batch);

	tcomplex* d_framesft;
	hipMalloc((void**)&d_framesft, ElementsFFT2(dims) * nframes * batch * sizeof(tcomplex));
	tcomplex* d_sumsft;
	hipMalloc((void**)&d_sumsft, ElementsFFT2(dims) * batch * sizeof(tcomplex));

	long batchsize = tmax(1, (1 << 28) / (ElementsFFT2(dims) * sizeof(tcomplex)));
	for (int b = 0; b < batch; b += batchsize)
	{
		uint curbatch = tmin(batch - b, batchsize);
		d_FFTR2C(d_frames + Elements2(dims) * b, d_framesft + ElementsFFT2(dims) * b, 2, toInt3(dims), curbatch);
	}

	d_ComplexMultiplyByVector(d_framesft, d_framespectra, d_framesft, ElementsFFT2(dims) * nframes * batch);
	d_ReduceAdd(d_framesft, d_sumsft, ElementsFFT2(dims), nframes, batch);

	d_ComplexDivideByVector(d_sumsft, d_sumspectra, d_sumsft, ElementsFFT2(dims) * batch);

	hipFree(d_framesft);
	hipFree(d_framesft);
}